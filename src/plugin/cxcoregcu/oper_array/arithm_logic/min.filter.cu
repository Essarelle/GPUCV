#include "hip/hip_runtime.h"
//CVG_LicenseBegin========================================== ====================
//
//	Copyright@ Institut TELECOM 2005
//		http://www.institut-telecom.fr/en_accueil.html 
//	
//	This software is a GPU accelerated library for computer-vision. It 
//	supports an OPENCV-like extensible interface for easily porting OPENCV 
//	applications.   
//	    
//	Contacts :
//		patrick.horain@it-sudparis.eu  
//		gpucv-developers@picoforge.int-evry.fr
//	
//	Project's Home Page :
//		https://picoforge.int-evry.fr/cgi-bin/twiki/view/Gpucv/Web/WebHome
//	
//	This software is governed by the CeCILL-B license under French law and
//	abiding by the rules of distribution of free software.  You can  use, 
//	modify and/ or redistribute the software under the terms of the CeCILL-B
//	license as circulated by CEA, CNRS and INRIA at the following URL
//	"http://www.cecill.info/licences/Licence_CeCILL-B_V1-en.html". 
//	
//================================================================CVG_LicenseEnd
#include <cxcoregcu/oper_array/arithm_logic/arithm_logic.h>

#if 1//_GPUCV_COMPILE_CUDA

_GPUCV_CXCOREGCU_EXPORT_CU
void gcuMin(CvArr* src1,CvArr* src2, CvArr* dst)
{
	CudaArithm_SwitchCHANNELS<KERNEL_ARITHM_OPER_MIN, GCUArithmStruct>(&varLocalArithm, src1, src2, dst, NULL, 1.);
}



_GPUCV_CXCOREGCU_EXPORT_CU
void gcuMinS(CvArr* src1,double val, CvArr* dst)
{
	float4 TempScalar;
	TempScalar.x = val;
	TempScalar.y = val;
	TempScalar.z = val;
	TempScalar.w = val;
	CudaArithm_SwitchCHANNELS<KERNEL_ARITHM_OPER_MIN, GCUArithmStruct>(&varLocalArithm, src1, NULL, dst, NULL, 1., &TempScalar);
}
#endif