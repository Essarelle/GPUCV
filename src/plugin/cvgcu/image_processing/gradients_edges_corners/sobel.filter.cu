//CVG_LicenseBegin==============================================================
//
//	Copyright@ Institut TELECOM 2005
//		http://www.institut-telecom.fr/en_accueil.html
//	
//	This software is a GPU accelerated library for computer-vision. It 
//	supports an OPENCV-like extensible interface for easily porting OPENCV 
//	applications.
//	
//	Contacts :
//		patrick.horain@it-sudparis.eu
//		gpucv-developers@picoforge.int-evry.fr
//	
//	Project's Home Page :
//		https://picoforge.int-evry.fr/cgi-bin/twiki/view/Gpucv/Web/WebHome
//	
//	This software is governed by the CeCILL-B license under French law and
//	abiding by the rules of distribution of free software.  You can  use, 
//	modify and/ or redistribute the software under the terms of the CeCILL-B
//	license as circulated by CEA, CNRS and INRIA at the following URL
//	"http://www.cecill.info/licences/Licence_CeCILL-B_V1-en.html". 
//	
//================================================================CVG_LicenseEnd



//Other Licenses:
//Some operators are inspired from the CUDA SDK, see corresponding license terms.
//	CUDA SDK:
//	- CudaSobelIpl() based on Sobel.
//==============================================================================
#include <cvgcu/config.h>
#include <GPUCVCuda/gpucv_wrapper_c.h>

#if _GPUCV_COMPILE_CUDA
#include <GPUCVCuda/base_kernels/tpl_convolutions.kernels.h>
#include <assert.h>
#include <cutil_inline.h>
#include <cvgcu/image_processing/filter_color_conv/cvgcu_convol_filter.kernel.h>
////////////////////////////////////////////////////////////////////////////////
// Convolution kernel storage
////////////////////////////////////////////////////////////////////////////////
//__constant__ unsigned char c_Kernel[KERNEL_LENGTH];


template <typename TPLSrc, typename TPLDst>
void convolutionRowsGPU(
    TPLSrc *d_Src,
    TPLDst *d_Dst,
    int imageW,
    int imageH
){
    assert( ROWS_BLOCKDIM_X * ROWS_HALO_STEPS >= KERNEL_RADIUS );
    assert( imageW % (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X) == 0 );
    assert( imageH % ROWS_BLOCKDIM_Y == 0 );

    dim3 blocks(imageW / (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X), imageH / ROWS_BLOCKDIM_Y);
    dim3 threads(ROWS_BLOCKDIM_X, ROWS_BLOCKDIM_Y);

    convolutionRowsKernel<<<blocks, threads>>>(
        d_Src,
		d_Dst,
        imageW,
        imageH,
        imageW
    );
    cutilCheckMsg("convolutionRowsKernel() execution failed\n");
} 

template <typename TPLSrc, typename TPLDst>
void convolutionColumnsGPU(
    TPLSrc *d_Src,
    TPLDst *d_Dst,
    int imageW,
    int imageH
){
    assert( COLUMNS_BLOCKDIM_Y * COLUMNS_HALO_STEPS >= KERNEL_RADIUS );
    assert( imageW % COLUMNS_BLOCKDIM_X == 0 );
    assert( imageH % (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y) == 0 );

    dim3 blocks(imageW / COLUMNS_BLOCKDIM_X, imageH / (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y));
    dim3 threads(COLUMNS_BLOCKDIM_X, COLUMNS_BLOCKDIM_Y);

    convolutionColumnsKernel<<<blocks, threads>>>(
        d_Src,
		d_Dst,
        imageW,
        imageH,
        imageW
    );
    cutilCheckMsg("convolutionColumnsKernel() execution failed\n");
}



/**
\return false if convolution kernel has not been calculated
\param src => Kernel will be faster if type is 32S/32U/32F
\param dst => type must be 16S/16U/32S/32U/32F.
\note In fact, all type are supported but results and performances are unknown for 8U/8S/...
*/
_GPUCV_CVGCU_EXPORT_CU
bool gcuSobel(void* src, void* dst, int xorder, int yorder, int aperture_size)
{
	unsigned int width		= gcuGetWidth(dst);
	unsigned int height		= gcuGetHeight(dst);
	
	//prepare input/ouput========
	void* d_src	= gcuPreProcess(src, GCU_INPUT, hipMemoryTypeDevice);
	void* d_dst = gcuPreProcess(dst, GCU_OUTPUT, hipMemoryTypeDevice);
	
    float *h_Kernel_Horiz = (float *)malloc(KERNEL_LENGTH * sizeof(float));
	float *h_Kernel_Vert = (float *)malloc(KERNEL_LENGTH * sizeof(float));
	
	//use float temporary image
	void *d_Buffer = NULL;
	hipMalloc((void **)&d_Buffer , width * height * sizeof(float));
	//=====================

	if(aperture_size==-1)//Scharr filter
	{
		h_Kernel_Vert[0] = 3;
		h_Kernel_Vert[1] = 10;
		h_Kernel_Vert[2] = 3;

		h_Kernel_Horiz[0] = -1;
		h_Kernel_Horiz[1] = 0;
		h_Kernel_Horiz[2] = 1;	
	}
	else if ((xorder==0) && (yorder==1) && (aperture_size==3))
	{
		h_Kernel_Vert[0] = 1;
		h_Kernel_Vert[1] = 2;
		h_Kernel_Vert[2] = 1;

		h_Kernel_Horiz[0] = 1;
		h_Kernel_Horiz[1] = 0;
		h_Kernel_Horiz[2] = -1;
	}
	else if ((xorder==1) && (yorder==0) && (aperture_size==3))
	{
		h_Kernel_Vert[0] = 1;
		h_Kernel_Vert[1] = 0;
		h_Kernel_Vert[2] = -1;

		h_Kernel_Horiz[0] = 1;
		h_Kernel_Horiz[1] = 2;
		h_Kernel_Horiz[2] = 1;
	}
	else
	{
		cutilSafeCall( hipFree(d_Buffer ) );
		free(h_Kernel_Horiz);
		free(h_Kernel_Vert);

		//clean input/output
		gcuPostProcess(dst);
		gcuPostProcess(src);
		return false;
	}

	hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel_H), h_Kernel_Horiz, KERNEL_LENGTH * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel_V), h_Kernel_Vert, KERNEL_LENGTH * sizeof(float));


	#define GCU_SOBEL_SWITCH_FCT(CHANNELS, SRC_TYPE, DST_TYPE)\
		convolutionRowsGPU<SRC_TYPE, float>((SRC_TYPE*)d_src,(float*)d_Buffer,width,height);\
		convolutionColumnsGPU<float, DST_TYPE>((float*)d_Buffer,(DST_TYPE*)d_dst,width,height);

	//run kernels for any kind of format...some format might not be used, but it is easier to mange all of them with 
	//this macro than set them manually...
	GCU_MULTIPLEX_CONVERT_1CHANNELS_ALLSRCFORMAT(GCU_SOBEL_SWITCH_FCT, 1, gcuGetGLDepth(src),gcuGetGLDepth(dst));
	
	
	gcudaThreadSynchronize();
    cutilSafeCall( hipFree(d_Buffer ) );
    free(h_Kernel_Horiz);
	free(h_Kernel_Vert);

	//clean input/output
	gcuPostProcess(dst);
	gcuPostProcess(src);
	return true;
}
//=========================================================
#endif//_GPUCV_COMPILE_CUDA
